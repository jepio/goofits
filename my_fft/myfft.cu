#include "hip/hip_runtime.h"
/* 
 * Test of the CuFFT library. Calculating the absolute values of the complex
 * output can be done using a kernel or a thrust::transform operation.
 */

#include "myfft.h"

template <typename T> std::vector<T> read_data(std::string filename)
{
    std::vector<T> data;
    std::ifstream file;
    file.open(filename.c_str());
    T t, y;
    int i=0;
    while (file.good()){
        file >> t >> y;
        data.push_back(y);
        i++;
    }
    // Last element gets read twice this way. Remove it.
    data.pop_back();
    i--;
    std::cout << i <<std::endl; 
    file.close();
    return data;
}

__global__ void magnitude(hipfftComplex *in, float *out, size_t size)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < size){
        out[idx] = hipCabsf(in[idx]);
    }
}

#ifdef THRUST
struct complex_mag_functor : public thrust::unary_function<hipfftComplex, float>
{
    complex_mag_functor(){}

    __host__ __device__ float operator()(hipfftComplex in)
    {
        return hipCabsf(in);
    }
};
#endif

template <typename T> std::vector<T> fft_cuda(std::vector<T>& in)
{
    hipfftReal *d_in;
    size_t output_size = in.size()/2+1;
    // Copy input data to GPU
    checkCudaErrors(hipMalloc((void **)&d_in,sizeof(T)*in.size()));
    checkCudaErrors(hipMemcpy(d_in, &in[0], sizeof(T)*in.size(),hipMemcpyHostToDevice));
    // Allocate space for output on GPU
    hipfftComplex *d_out;
    checkCudaErrors(hipMalloc((void **)&d_out,sizeof(*d_out)*output_size));
    // Perform FFT
    hipfftHandle plan;
    hipfftPlan1d(&plan,in.size(), HIPFFT_R2C,1); 
    hipfftExecR2C(plan,d_in,d_out);
    // Calculate absolute values on GPU and copy to CPU
    std::vector<T> out;
    #ifndef THRUST
    T *d_abs;
    checkCudaErrors(hipMalloc((void **)&d_abs, sizeof(*d_abs)*output_size));
    size_t blockSize = 1024;
    size_t gridSize = output_size / blockSize + 1;
    magnitude <<<gridSize, blockSize>>>(d_out, d_abs, output_size);
    
    out.resize(output_size);
    checkCudaErrors(hipMemcpy(&out[0], d_abs,
                    sizeof(*d_abs)*output_size,hipMemcpyDeviceToHost));
    checkCudaErrors(hipDeviceSynchronize());
    #endif
    // Thrust version
    #ifdef THRUST
    thrust::device_ptr<hipfftComplex> dev_thr_out(d_out);
    out.resize(output_size);
    thrust::device_vector<T> thr_out(output_size);
    thrust::transform(dev_thr_out, dev_thr_out+output_size, thr_out.begin(),complex_mag_functor());
    thrust::copy(thr_out.begin(),thr_out.end(),&out[0]);
    hipfftDestroy(plan);
    #endif
    hipFree(d_in);
    hipFree(d_out);
    #ifndef THRUST
    hipFree(d_abs);
    #endif
    return out;
}

int main(void)
{
    std::vector<float> in;
    /*
     * Theoretically I used templates, however it will only work for floats.
     * Could be expanded to support doubles, but thats about all that can be
     * done.
     */
    in = read_data<float>("in.file");
    assert(in.size() != 0);
    std::vector<float> out;
    out = fft_cuda(in);
    std::ofstream outfile;
    outfile.open("fft.file");
    if (outfile.is_open()){
        for(unsigned int i=0;i<out.size();i++)
            outfile<<out[i]<<std::endl;
    }
    outfile.close();
    return 0;
}
